//**************************************************************************\
//* This file is property of and copyright by the ALICE Project            *\
//* ALICE Experiment at CERN, All rights reserved.                         *\
//*                                                                        *\
//* Primary Authors: Matthias Richter <Matthias.Richter@ift.uib.no>        *\
//*                  for The ALICE HLT Project.                            *\
//*                                                                        *\
//* Permission to use, copy, modify and distribute this software and its   *\
//* documentation strictly for non-commercial purposes is hereby granted   *\
//* without fee, provided that the above copyright notice appears in all   *\
//* copies and that both the copyright notice and this permission notice   *\
//* appear in the supporting documentation. The authors make no claims     *\
//* about the suitability of this software for any purpose. It is          *\
//* provided "as is" without express or implied warranty.                  *\
//**************************************************************************

/// \file GPUReconstructionCUDAExternalProvider.cu
/// \author David Rohr

#include "GPUReconstructionCUDADef.h"
#include "GPUReconstructionCUDAIncludes.h"

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "CUDAThrustHelpers.h"

#include <stdexcept>

using namespace GPUCA_NAMESPACE::gpu;

#include "GPUConstantMem.h"

// Files needed for O2 propagator
#include "MatLayerCylSet.cxx"
#include "MatLayerCyl.cxx"
#include "Ray.cxx"
#include "TrackParametrization.cxx"
#include "TrackParametrizationWithError.cxx"
#include "Propagator.cxx"
#include "TrackLTIntegral.cxx"

#ifndef GPUCA_NO_CONSTANT_MEMORY
static GPUReconstructionDeviceBase::deviceConstantMemRegistration registerConstSymbol([]() {
  void* retVal = nullptr;
  if (hipGetSymbolAddress(&retVal, HIP_SYMBOL(gGPUConstantMemBuffer)) != hipSuccess) {
    throw std::runtime_error("Could not obtain GPU constant memory symbol");
  }
  return retVal;
});
#endif
